#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include<hip/hip_runtime.h>                                                                                                                        
#include<hipsparse.h> 

/*cuSPARSE status and handle definition*/
hipsparseStatus_t status;
hipsparseHandle_t handle=0;

/*CUDA error wrapper*/
static void CUDA_ERROR( hipError_t err)
{
  if(err!= hipSuccess){
    printf("CUDA ERROR: %s, exiting\n", hipGetErrorString(err));
    exit(-1);
    }
}

/*Right and left hand side voltages*/
Scalar rhsV(Scalar t)
{
  return 0.;
}
Scalar lhsV(Scalar t)
{
  return lhsvoltage;
}


/*################################################################/*
 * Fields initializers *
 * ###############################################################*/

/*Array allocation in CPU and GPU*/
void allocate_arrays(int dealloc=0)
{
  if(dealloc)
    {
      /*Delete CPU arrays*/
      delete [] narray;
      delete [] phiarray;
      delete [] Earray;
      
      /*Free GPU memory*/
      hipFree(Earray_gpu);
      hipFree(narray_gpu);
      hipFree(nc_gpu);
      hipFree(d_a);
      hipFree(d_b);
      hipFree(d_c);
      return;
    }
  
  /*Allocate GPU matrix arrays*/ 
  hipMalloc((void**)&Earray_gpu, ng*sizeof(double));
  hipMalloc((void**)&narray_gpu, ng*sizeof(double));
  hipMalloc((void**)&nc_gpu, sizeof(int));
  hipMalloc((void **)&d_a, ng*sizeof(double));
  hipMalloc((void **)&d_b, ng*sizeof(double));
  hipMalloc((void **)&d_c, ng*sizeof(double));
  
  /*Create CPU arrays*/  
  narray = new double[ng];
  phiarray = new double[ng];
  Earray = new double[ng];
}

/*Sets the matrix coefficients in each thread*/
__global__ void setcoeffs(Scalar scale, double *a,double *b,double *c, int *nc_gpu)
{
  /*the id of the thread*/
  int tid=blockIdx.x*blockDim.x+threadIdx.x;           

  /*Each thread writes three values depending on its tid.
   a is lower diagonal, b diagonal, c upper diagonal*/
  if(tid == 0 )
  {
      a[tid] = 0.0;
      b[tid] = 1.0;
      c[tid] = 0.0;
	}
  else if (tid == *nc_gpu)
  {
      a[tid] = 0.0;
      b[tid] = 1.0;
      c[tid] = 0.0;
  } 
  else if( (tid>0)&&(tid<*nc_gpu))
  {
      a[tid] = scale;
      b[tid] = -2.*scale;
      c[tid] = scale;
  }
}

/*Initialization of all variables and arrays*/
void init_fields()
{
  /*Array allocation*/
  allocate_arrays();
  
  /*Setting initial values*/
  hipMemcpy(nc_gpu, &nc, sizeof(int), hipMemcpyHostToDevice);
  hipMemset(Earray_gpu, 0, ng*sizeof(double));
  
  /*Definition of GPU threads and blocks for the correct
   behaviour of the field solver. Nthreads_fields may vary 
   with hardware*/
  Nblocks_fields = ng/Nthreads_max +1;
  /*Set matrix coefficients*/
  setcoeffs<<<Nblocks_fields,Nthreads_max>>>(-epsilon/(q*dx*dx), d_a, d_b, d_c, nc_gpu);

  /*Create a cuSPARSE handle. Needed to call sparse functions*/
  status=hipsparseCreate(&handle);
}

/*################################################################/*
 * Laplace and gradient functions *
 * ###############################################################*/

/*Sums the Laplace equation for rhs and lhs voltages*/
__global__ void sumLaplace_GPU(double *pphh, Scalar dx, Scalar rv, Scalar lv, Scalar xl, Scalar L, int ng)
{
  int tid=blockIdx.x*blockDim.x+threadIdx.x;           
  Scalar frac, xlocal;

  if( (tid >= 0) && ( tid < ng)){
    xlocal = xl + tid*dx;
    frac = xlocal/L;
    pphh[tid] += (frac*rv + (1. - frac)*lv) ;
  }
}

/*Computes the gradient in GPU*/
__global__ void gradient_GPU(double *grad, double *arr, int n, Scalar scale)
{
  int tid=blockIdx.x*blockDim.x+threadIdx.x;           

  /*Intermediate points calculation*/
  if( (tid>0) && (tid < n))
  {
    grad[tid] = scale*(arr[tid+1] - arr[tid-1]);
  }
  
  /*Fix up end points*/
  if(tid==0)
  {
    grad[0] = 2*scale*(arr[1] - arr[0]);
  }
  if(tid==n)
  {
    grad[n] = 2*scale*(arr[n] - arr[n-1]);
  }
}

/*################################################################/*
 * Main function *
 * ###############################################################*/

/*Updates phi and E values in dt*/
void advancefields(Scalar ddt)
{
  starttime(FIELDS);

  /*Correct density array for computation*/
  starttime(MEMORY_GPU);
  hipMemset(&(narray_gpu[0]), 0, sizeof(double)); 
  hipMemset(&(narray_gpu[nc]), 0, sizeof(double)); 
  endtime(MEMORY_GPU);

  starttime(PERFORMANCE);
  /*Trimatrix solver
   *Function from cuSPARSE for trimatrix solution*
   *Overwrites n_array with phi values*/
  status=cusparseDgtsv(handle,nc,1,d_a,d_b,d_c,narray_gpu,nc);
  if (status != HIPSPARSE_STATUS_SUCCESS)
  {
     std::cout << status << std::endl;
  }  
  endtime(PERFORMANCE);

  /*Laplace correction and gradient calculation with values
   * that are already in GPU*/
  sumLaplace_GPU<<<Nblocks_fields,Nthreads_max>>>(narray_gpu, dx, rhsV(t), lhsV(t), xl, L, ng);
  gradient_GPU<<<Nblocks_fields,Nthreads_max>>>( Earray_gpu,  narray_gpu,  nc,  -0.5/dx);
  
  /*Copy data back to CPU*/
  if(diag_flag){
    starttime(MEMORY_GPU);
    hipMemcpy(phiarray, narray_gpu, ng*sizeof(double),hipMemcpyDeviceToHost);
    hipMemcpy(Earray, Earray_gpu,ng*sizeof(double),hipMemcpyDeviceToHost);
    endtime(MEMORY_GPU);
  }
  
  endtime(FIELDS);
}
